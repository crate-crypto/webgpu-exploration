#include "hip/hip_runtime.h"
/***

Copyright (c) 2022, Yrrid Software, Inc.  All rights reserved.
Licensed under the Apache License, Version 2.0, see LICENSE for details.

Written by Niall Emmart.

***/

#include <stdio.h>
#include <stdint.h>

#include "MSM.h"

#include "asm.cu"
#include "Support.cu"
#include "Chain.cu"
#include "MP.cu"
#include "Curve.cu"
#include "SHM.cu"

#include "PrecomputePoints.cu"
#include "ProcessSignedDigits.cu"
#include "InitializeCountersAndSizes.cu"
#include "Partition1024.cu"
#include "Partition4096.cu"
#include "SortCounts.cu"

#include "ComputeBucketSums.cu"
#include "ReduceBuckets.cu"

#include "HostCurve.cpp"
#include "HostReduce.cpp"

#define ROUND128(x) (x + 127 & 0xFFFFFF80)
#define ROUND256(x) (x + 255 & 0xFFFFFF00)

#define CUDA_CHECK(call) if((errorState=call)!=0) { hipError_t("Call \"" #call "\" failed.", __FILE__, __LINE__); return errorState; }

uint32_t MAX128(uint32_t a, uint32_t b) {
  return ROUND128(a>=b ? a : b);
}

uint32_t MAX128(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t max;
  
  max=(a>=b ? a : b);
  max=(max>=c ? max : c);
  return ROUND128(max);
}

void* advanceBytes(void*& current, uint32_t bytes) {
  uint8_t* prior=(uint8_t*)current;
  
  current=(void*)(prior + bytes); 
  return (void*)prior;
}

void* advanceScalars(void* scalars, uint32_t count) {
  uint8_t* ptr8=(uint8_t*)scalars;
  
  return (void*)(ptr8 + ((uint64_t)32)*((uint64_t)count));
}

void* advanceFields(void* results, uint32_t fieldCount) {
  uint8_t* ptr8=(uint8_t*)results;
  
  return (void*)(ptr8 + fieldCount*48);
}

#if defined(SUPPORT_READING)

#include "Reader.cpp"

// C interface implementations
int32_t MSMReadHexPoints(uint8_t* pointsPtr, uint32_t count, const char* path) {
  FILE* f=fopen(path, "r");
  
  // absolutely horrible, but I don't know how to load points in Rust
  
  if(f==NULL) {
    fprintf(stderr, "Failed to open path '%s' for reading\n", path);
    return -1;
  }
  
  for(uint32_t i=0;i<count;i++) {
    for(uint32_t j=0;j<2;j++) {
      if(!parseHex(pointsPtr + j*48, f, 48)) {
        fprintf(stderr, "Points file parse failed\n");
        return -1;
      }
    }
    for(uint32_t j=0;j<8;j++)
      pointsPtr[j + 96]=0;     
    pointsPtr+=104;
  }
  
  fclose(f);
  return 0;
}

int32_t MSMReadHexScalars(uint8_t* scalarsPtr, uint32_t count, const char* path) {
  FILE* f=fopen(path, "r");
  
  // absolutely horrible, but I don't know how to load points in Rust
  
  if(f==NULL) {
    fprintf(stderr, "Failed to open path '%s' for reading\n", path);
    return -1;
  }
  
  for(uint32_t i=0;i<count;i++) {
    if(!parseHex(scalarsPtr, f, 32)) {
      fprintf(stderr, "Scalar file parse failed\n");
      return -1;
    }
    scalarsPtr += 32;
  }
  
  fclose(f);
  return 0;
}

#endif

void* MSMAllocContext(int32_t maxPoints, int32_t maxBatches) {
  return (void*)(new MSMContext(maxPoints, maxBatches));
}

int32_t MSMFreeContext(void* context) {
  delete (MSMContext*)context;
  return 0;
}

int32_t MSMPreprocessPoints(void* context, void* affinePointsPtr, uint32_t points) {
  return ((MSMContext*)context)->msmPreprocessPoints(affinePointsPtr, points);
}

int32_t MSMRun(void* context, uint64_t* projectiveResultsPtr, void* scalarsPtr, uint32_t scalars) {
  return ((MSMContext*)context)->msmRun(projectiveResultsPtr, scalarsPtr, scalars);
}

MSMContext::MSMContext(uint32_t _maxPoints, uint32_t _maxBatches) {
  maxPoints=ROUND256(_maxPoints);
  maxBatches=_maxBatches;
  smCount=0;
  errorState=0;
  gpuPlanningMemory=NULL;
  gpuPointsMemory=NULL;
  cpuReduceResults=NULL;
}

MSMContext::~MSMContext() {
  // FIX FIX FIX!  Need to clean up streams and events
  
  if(gpuPlanningMemory!=NULL)
    hipFree(gpuPlanningMemory);
  if(gpuPointsMemory!=NULL)
    hipFree(gpuPointsMemory);
  if(cpuReduceResults!=NULL)
    hipHostFree(cpuReduceResults);
}

void MSMContext::hipError_t(const char* call, const char* file, uint32_t line) {
  fprintf(stderr, "CUDA Error %d occurred on \"%s\", in %s:%d\n", errorState, call, file, line);
}

size_t MSMContext::memoryLayoutSize() {
  size_t    totalBytes;
  uint32_t  counters, pointsPerPage, pageCount, sizeCount;
  uint32_t  overlay1a, overlay1b, overlay2a, overlay3a, overlay3b, overlay3c;

  pointsPerPage=(PAGE_SIZE-4)/5;
  counters=11*1024 + 128;
  sizeCount=11*1024;
  pageCount=(maxPoints*11 + pointsPerPage - 1)/pointsPerPage + 11*1024; 
     
  // Processing performed:
  //   scalars -> processedScalars            
  //   processedScalars -> pages              
  //   pages -> points + unsortedTriple (uses scratch)  
  //   unsortedTriple -> sortedTriple
  ///  points + sortedTriple -> buckets
   
  //  Overlay 1:   scalars / pages
  //  Overlay 2:   points + unsorted triple
  //  Overlay 3:   processedScalars / scratch / sortedTriple 
  //  Overlay 4:   buckets
  //  Overlay 5:   misc
  
  // hopefully none of these exceed 2^32
  overlay1a=maxPoints*32;
  overlay1b=pageCount*PAGE_SIZE;
  ml.overlay1=MAX128(overlay1a, overlay1b);
 
  overlay2a=maxPoints*44;
  overlay2a+=NBUCKETS*(11+11)*4;
  ml.overlay2=overlay2a;
  
  overlay3a = maxPoints*33;
  overlay3b = smCount*ROUND128(SCRATCH_REQUIRED);
  overlay3c = NBUCKETS*(2+12+12)*4 + 32*(1+6+6)*4;
  ml.overlay3 = MAX128(overlay3a, overlay3b, overlay3c);
    
  ml.overlay4 = (NBUCKETS+NBUCKETS+32)*192;
  
  ml.overlay5 = ROUND128(128*4 + counters*8 + sizeCount*4 + sizeCount*4 + 1024*4 + smCount*8*3*192*maxBatches);
  
  totalBytes=ml.overlay1; 
  totalBytes+=ml.overlay2;
  totalBytes+=ml.overlay3;
  totalBytes+=ml.overlay4;
  totalBytes+=ml.overlay5;

  if(totalBytes<104ull * maxPoints)
    totalBytes=104ull * maxPoints;
  return totalBytes;
}

int32_t MSMContext::initializeMemoryLayout() {
  uint32_t  counters, pointsPerPage, pageCount, sizeCount;
  void*     overlay=gpuPlanningMemory;
  void*     current;
  
  pointsPerPage=(PAGE_SIZE-4)/5;
  counters=11*1024 + 128;
  sizeCount=11*1024;
  pageCount=(maxPoints*11 + pointsPerPage - 1)/pointsPerPage + 11*1024; 

  // Processing performed:
  //   scalars -> processedScalars            
  //   processedScalars -> pages              
  //   pages -> points + unsortedTriple (uses scratch)  
  //   unsortedTriple + points -> sortedTriple
  ///  points + sortedTriple -> buckets
   
  //  Overlay 1:   scalars / pages
  //  Overlay 2:   points + unsorted triple
  //  Overlay 3:   processedScalars / scratch / sortedTriple
  //  Overlay 4:   buckets
  //  Overlay 5:   misc

  // OVERLAY 1
    current=overlay;
    ml.scalars=advanceBytes(current, maxPoints*32);

    current=overlay;
    ml.pages=advanceBytes(current, pageCount*PAGE_SIZE);
 
    advanceBytes(overlay, ml.overlay1);
  
  // OVERLAY 2
    current=overlay;
    ml.points=advanceBytes(current, maxPoints*44);
    ml.unsortedTriple=advanceBytes(current, NBUCKETS*(11+11)*4);
    
    advanceBytes(overlay, ml.overlay2);
  
  // OVERLAY 3
    current=overlay;
    ml.processedScalars=advanceBytes(current, maxPoints*33);
  
    current=overlay;
    ml.scratch=advanceBytes(current, smCount*ROUND128(SCRATCH_REQUIRED));
     
    current=overlay;
    ml.sortedTriple=advanceBytes(current, NBUCKETS*(2+12+12)*4 + 32*(1+6+6)*4);

    advanceBytes(overlay, ml.overlay3);

  // OVERLAY 4 
    current=overlay;
    ml.buckets=advanceBytes(current, (NBUCKETS+NBUCKETS+32)*192);
    
    advanceBytes(overlay, ml.overlay4);
    
  // OVERLAY 5 
    current=overlay;
    ml.atomics=advanceBytes(current, 128*4);
    ml.counters=advanceBytes(current, counters*8);
    ml.sizes=advanceBytes(current, sizeCount*4);
    ml.prefixSumSizes=advanceBytes(current, sizeCount*4);
    ml.histogram=advanceBytes(current, 1024*4);
    ml.results=advanceBytes(current, smCount*8*3*192*maxBatches);
  
    advanceBytes(overlay, ml.overlay5);

  return 0;  
}
  
int32_t MSMContext::initializeGPU() {
  hipDeviceProp_t properties;
  
  if(errorState!=0)
    return errorState;
  
  if(smCount!=0) {
    // we're already initialized
    return 0;
  }

  CUDA_CHECK(hipGetDeviceProperties(&properties, 0));

  smCount=properties.multiProcessorCount;
  
  CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(partition1024Kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 64*1024));
  CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(partition4096Kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 64*1024));
  CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(sortCountsKernel), hipFuncAttributeMaxDynamicSharedMemorySize, 96*1024));
  CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(computeBucketSums), hipFuncAttributeMaxDynamicSharedMemorySize, 96*1024));
  
  CUDA_CHECK(hipStreamCreate(&runStream));
  CUDA_CHECK(hipStreamCreate(&memoryStream));
  
  CUDA_CHECK(hipEventCreate(&planningComplete));
  CUDA_CHECK(hipEventCreate(&lastRoundPlanningComplete));
  CUDA_CHECK(hipEventCreate(&writeComplete));
  
  CUDA_CHECK(hipEventCreate(&timer0));
  CUDA_CHECK(hipEventCreate(&timer1));
  CUDA_CHECK(hipEventCreate(&timer2));
  CUDA_CHECK(hipEventCreate(&timer3));
  CUDA_CHECK(hipEventCreate(&timer4));
    
  CUDA_CHECK(hipMalloc(&gpuPlanningMemory, memoryLayoutSize()));

#if defined(SMALL)
  CUDA_CHECK(hipMalloc(&gpuPointsMemory, 96ull * 6 * 65536));
#else
  CUDA_CHECK(hipMalloc(&gpuPointsMemory, 96ull * 6 * maxPoints));
#endif

  if(initializeMemoryLayout()!=0)
    return errorState;

  CUDA_CHECK(hipHostMalloc((void**)&cpuReduceResults, maxBatches*smCount*8*3*192));
  return 0;
}

void MSMContext::hostReduce(uint64_t* projectiveResultsPtr, uint32_t batch) {
  typedef Host::BLS12377::G1Montgomery Field;
  typedef Host::HostReduce<Field>      HostReduce;

  HostReduce hostReduce(2, 23, smCount*8);

  hostReduce.reduce(projectiveResultsPtr + batch*6*3, (uint32_t*)advanceFields(cpuReduceResults, batch*smCount*8*3*4));   
}

int32_t MSMContext::msmPreprocessPoints(void* affinePointsPtr, uint32_t points) {
  uint32_t basePoints;
  
  if(errorState!=0)
    return errorState;

  if(initializeGPU()<0)
    return errorState;

  if(points>maxPoints) {
    fprintf(stderr, "Point count exceeded max points\n");
    return -1;
  }
  
  if(points%65536!=0) {
    fprintf(stderr, "Point count must be evenly divisible by 65536\n");
    return -1;
  }
  
  #if defined(SMALL)
    basePoints=65536;
  #else
    basePoints=points;  
  #endif
  
  // This kernel pre-computes the following for each input point, Pi:
  //   2^46*Pi, 2^92*Pi, 2^138*Pi, 2^184*Pi, 2^230*Pi, 2^276*Pi
  // These precomputed points let us reduce the computation from 11 window @ 23 bits per window
  // down to 2 windows.  This pre-computation is part of the setup routine and is not timed.
  
  CUDA_CHECK(hipMemcpy(gpuPlanningMemory, affinePointsPtr, ((uint64_t)104)*((uint64_t)basePoints), hipMemcpyHostToDevice));
  precomputePointsKernel<<<smCount, 256, 1536>>>(gpuPointsMemory, gpuPlanningMemory, basePoints);
  CUDA_CHECK(hipDeviceSynchronize());

  preprocessedPoints=points;
  
  return 0;
}

int32_t MSMContext::msmRun(uint64_t* projectiveResultsPtr, void* scalarsPtr,  uint32_t scalars) {
  uint32_t  points=preprocessedPoints, batches=scalars/points;
  void*     partition1024Args[5]={&ml.pages, &ml.sizes, &ml.counters, &ml.processedScalars, &points};
  void*     sizesPrefixSumArgs[5]={&ml.pages, &ml.prefixSumSizes, &ml.sizes, &ml.counters, &ml.atomics};
  void*     nextScalarsPtr=scalarsPtr;
  void*     nextResultsPtr=ml.results;
  
  if(errorState!=0)
    return errorState;
    
  if(scalars%points!=0) {
    fprintf(stderr, "Scalar count must be a multiply of point count\n");
    return -1;
  }
  
  if(batches>maxBatches) {
    fprintf(stderr, "Batch count exceed max batches\n");
    return -1;
  }
  
  if(preprocessedPoints!=points) {
    fprintf(stderr, "Points count does not match preprocessed points\n");
    return -1;
  }
  
  // The initial copy time of 2^26 scalars is pretty slow.
  
  CUDA_CHECK(hipEventRecord(timer0, runStream));  
  CUDA_CHECK(hipMemcpy(ml.scalars, nextScalarsPtr, points*32u, hipMemcpyHostToDevice));
  nextScalarsPtr=advanceScalars(nextScalarsPtr, points);
  CUDA_CHECK(hipEventRecord(timer1, runStream));  

  for(uint32_t batch=1;batch<=batches;batch++) {    
    if(batch>0)
      CUDA_CHECK(hipStreamWaitEvent(runStream, writeComplete));

    // This kernel breaks each scalar value into 11 windows, and does signed-digit processing.  The window value
    // has a sign bit and a 22-bit scalar value (the Pippenger bucket).  Note, 23 evenly divides 253, so use a 
    // small trick -- if the high bit of the scalar is set, we negate the point and change the scalar to
    //    s'=m - s
    // where m is the order of the field.  The new scalar, s', will have the high bit clear.   This works since:
    //    s' (-Pi) = (m - s) (-Pi) = -s -Pi = s Pi.
    
    processSignedDigitsKernel<<<points/256, 256, 8928, runStream>>>(ml.processedScalars, ml.scalars, points);
 
    // These next 6 kernels are a replacement for CUB sorting.  Their implementation is involved, but conceptually 
    // what they do is quite simple:  they group all the points together that need to go into the same bucket.  What
    // comes is 11x 2^22 lists of points.  The buckets are then sorted such that buckets that will accumulate the most
    // points are processed first.
  
    initializeCountersSizesAtomicsHistogramKernel<<<smCount, 256, 0, runStream>>>(ml.counters, ml.sizes, ml.atomics, ml.histogram);
    CUDA_CHECK(hipLaunchCooperativeKernel((const void*)partition1024Kernel, dim3(smCount), dim3(1024), partition1024Args, 64*1024, runStream));
    CUDA_CHECK(hipLaunchCooperativeKernel((const void*)sizesPrefixSumKernel, dim3(11), dim3(1024), sizesPrefixSumArgs, 0, runStream));
    partition4096Kernel<<<smCount, 1024, 64*1024, runStream>>>(ml.points, ml.unsortedTriple, ml.scratch, ml.prefixSumSizes, ml.sizes, ml.pages, ml.atomics, points);
    histogramPrefixSumKernel<<<smCount, 1024, 0, runStream>>>(ml.histogram, ml.unsortedTriple);
    sortCountsKernel<<<smCount, 1024, 96*1024, runStream>>>(ml.sortedTriple, ml.histogram, ml.unsortedTriple);

    if(batch!=batches) {
      // DO NOT REMOVE BRACKETS
      CUDA_CHECK(hipEventRecord(planningComplete, runStream));
    }
    else {
      // DO NOT REMOVE BRACKETS
      CUDA_CHECK(hipEventRecord(lastRoundPlanningComplete, runStream));
    }

    // ComputeBucketSums processes the lists, and computes a bucket sum for each list.  The kernel works by assigning a
    // thread to each bucket, and uses an EC add routines, based on XYZZ representation.  Since the buckets have been sorted 
    // by the number of points in each bucket, it's almost always the case that all the threads in each warp and converged 
    // and we can take advantage of copying for the first point in a bucket and use the faster "zz=1/zzz=1" addition
    // for the second point in each bucket.  Finally, again because the buckets are converged, all threads in the
    // warp write their results to global memory at the same time.
    // ReduceBuckets launches smCount*8 warps.  Half the warps are used to reduce each window.  

    computeBucketSums<<<smCount, 384, 96*1024, runStream>>>(ml.buckets, gpuPointsMemory, ml.sortedTriple, ml.points, ml.atomics);
    reduceBuckets<<<smCount, 256, 256*96 + 1536, runStream>>>(nextResultsPtr, ml.buckets);
    nextResultsPtr=advanceFields(nextResultsPtr, smCount*8*3*4);
    
    if(batch!=batches) {  
      CUDA_CHECK(hipStreamWaitEvent(memoryStream, planningComplete))
      if(batch==0) {
        CUDA_CHECK(hipMemcpyAsync(ml.scalars, nextScalarsPtr, (points-points/4)*32u, hipMemcpyHostToDevice, memoryStream));
        nextScalarsPtr=advanceScalars(nextScalarsPtr, points-points/4);
      }
      else {
        CUDA_CHECK(hipMemcpyAsync(ml.scalars, nextScalarsPtr, points*32u, hipMemcpyHostToDevice, memoryStream));
        nextScalarsPtr=advanceScalars(nextScalarsPtr, points);
      }
      CUDA_CHECK(hipEventRecord(writeComplete, memoryStream));
    }
  }

  CUDA_CHECK(hipEventSynchronize(lastRoundPlanningComplete));
  CUDA_CHECK(hipMemcpyAsync(cpuReduceResults, ml.results, (batches-1)*smCount*8*3*192, hipMemcpyDeviceToHost, memoryStream));
  CUDA_CHECK(hipStreamSynchronize(memoryStream));

  // all but the last one
  for(uint32_t batch=0;batch<batches-1;batch++)
    hostReduce(projectiveResultsPtr, batch);
  
  CUDA_CHECK(hipStreamSynchronize(runStream));
  CUDA_CHECK(hipMemcpy(advanceFields(cpuReduceResults, (batches-1)*smCount*8*3*4),
                        advanceFields(ml.results, (batches-1)*smCount*8*3*4),
                        smCount*8*3*192, hipMemcpyDeviceToHost));
  CUDA_CHECK(hipEventRecord(timer2, 0));

  // reduce the very last result
  hostReduce(projectiveResultsPtr, batches-1);

  float ms;
  hipEventElapsedTime(&ms, timer0, timer1);
  printf("Initial copy: %f ms\n", ms);
  hipEventElapsedTime(&ms, timer0, timer2);
  printf("Total time: %f ms\n", ms);
  
  return 0;
}

int main() {
  MSMContext context(100000, 100000);

  uint32_t affinePointsPtr[24] = {0, 1,2,3,4,5,6,70,8,9,10,11, 0, 1,2,3,4,5,6,70,8,9,10,11};
  uint64_t projectiveResultsPtr[48] = {7952349086523354480, 16914211206557533, 1804455504723096531, 0, 0, 0, 7952349086523354480, 16914211206557533, 1804455504723096531, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  uint32_t scalarsPtr[32] = {3697582722, 2119299629, 2515941055, 2806193226, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

  context.msmPreprocessPoints(affinePointsPtr, 65536);
  context.msmRun(projectiveResultsPtr, scalarsPtr, 65536);
}
