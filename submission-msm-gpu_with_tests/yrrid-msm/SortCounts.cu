#include "hip/hip_runtime.h"
/***

Copyright (c) 2022, Yrrid Software, Inc.  All rights reserved.
Licensed under the Apache License, Version 2.0, see LICENSE for details.

Written by Niall Emmart.

***/


__global__ void histogramPrefixSumKernel(void* histogramPtr, void* unsortedTriplePtr) {
  uint32_t  globalTID=blockIdx.x*blockDim.x+threadIdx.x, globalStride=gridDim.x*blockDim.x;
  uint32_t* histogram=(uint32_t*)histogramPtr;
  uint32_t* counts=(uint32_t*)unsortedTriplePtr;
  uint32_t  count, localSum;
  uint32_t  i;
  
  __shared__ uint32_t sharedHistogram[1024];
  __shared__ uint32_t warpTotals[32];
  
  // must launch with 1024 threads
  
  sharedHistogram[threadIdx.x]=0;

  __syncthreads();
    
  #pragma unroll 1
  for(i=globalTID;i<NBUCKETS;i+=globalStride) {
    count=0;
    #pragma unroll
    for(int32_t j=0;j<=10*NBUCKETS;j+=2*NBUCKETS) 
      count+=counts[j + i];
    count=umin(count, 1023);
    atomicAdd(&sharedHistogram[1023-count], 1);
  }
  
  #pragma unroll 1
  for(;i<2*NBUCKETS;i+=globalStride) {
    count=0;
    #pragma unroll
    for(int32_t j=0;j<=8*NBUCKETS;j+=2*NBUCKETS) 
      count+=counts[j + i];
    count=umin(count, 1023);
    atomicAdd(&sharedHistogram[1023-count], 1);
  }
  
  __syncthreads();

  count=sharedHistogram[threadIdx.x];
  localSum=multiwarpPrefixSum(warpTotals, count, 32);
  atomicAdd(&histogram[threadIdx.x], localSum-count);
}

__global__ void sortCountsKernel(void* sortedTriplePtr, void* histogramPtr, void* unsortedTriplePtr) {
  uint32_t  globalTID=blockIdx.x*blockDim.x+threadIdx.x, globalStride=gridDim.x*blockDim.x;
  uint32_t  warp=threadIdx.x>>5, warpThread=threadIdx.x & 0x1F, warps=blockDim.x>>5;
  uint32_t* histogram=(uint32_t*)histogramPtr;
  
  uint32_t  counts[6], indexes[6];
  uint32_t  count, bin, binCount, writeIndex, mask, thread, localWriteIndex, localBin, localBucket;
  bool      processed;

  // input pointers
  uint32_t* unsortedCounts=(uint32_t*)unsortedTriplePtr;
  uint32_t* unsortedIndexes=((uint32_t*)unsortedTriplePtr) + NBUCKETS*11;
  
  // output pointers
  uint32_t* sortedBuckets=(uint32_t*)sortedTriplePtr;
  uint4*    sortedCountsAndIndexes=(uint4*)(sortedBuckets + NBUCKETS*2 + 32);
  //uint32_t* sortedCounts=((uint32_t*)sortedTriplePtr) + NBUCKETS*2 + 32;
  //uint32_t* sortedIndexes=((uint32_t*)sortedTriplePtr) + NBUCKETS*14 + 32*7;
  
  extern __shared__ uint32_t shmem[];
  
  uint32_t* binCounts=shmem;                         // 1*256 (words)
  uint32_t* buckets=shmem+256;                       // 7*256 (words)
  uint4*    countsAndIndexes=(uint4*)(shmem+8*256);  // 7*12*256

  if(globalTID<384) {
    // 32 empty entries
    if(globalTID<32)
      sortedBuckets[NBUCKETS*2 + globalTID]=NBUCKETS*2 + globalTID;

    // 32*12 words: counts and indexes
    sortedBuckets[NBUCKETS*26 + globalTID + 32]=0;
  }

  for(int32_t i=threadIdx.x;i<256;i+=blockDim.x) 
    binCounts[i]=0;
  
  for(int32_t i=threadIdx.x;i<7*256;i+=blockDim.x) 
    buckets[i]=0xFFFFFFFF;
    
  __syncthreads();
  
  #pragma unroll 1
  for(uint32_t bucket=globalTID;bucket<2*NBUCKETS;bucket+=globalStride) {
    // collect the data
    if(bucket<NBUCKETS) {
      count=0;
      #pragma unroll
      for(int32_t i=0;i<6;i++) {
        counts[i]=unsortedCounts[NBUCKETS*2*i + bucket];
        indexes[i]=unsortedIndexes[NBUCKETS*2*i + bucket];
        count+=counts[i];
      }
    }
    else {
      count=0;
      #pragma unroll
      for(int32_t i=0;i<5;i++) {
        counts[i]=unsortedCounts[NBUCKETS*2*i + bucket];
        indexes[i]=unsortedIndexes[NBUCKETS*2*i + bucket];
        count+=counts[i];
      }
      counts[5]=0;
      indexes[5]=0;
    }
    
    processed=count>255;

    // if we have a lot of points in the coalesced bucket, do special one-off processing
    if(processed) {
      bin=umax(count, 1023);
      writeIndex=atomicAdd(&histogram[1023-bin], 1);
      sortedBuckets[writeIndex]=bucket;
      #pragma unroll
      for(int i=0;i<3;i++)
        sortedCountsAndIndexes[writeIndex*3 + i]=make_uint4(counts[i*2 + 0], indexes[i*2 + 0], counts[i*2 + 1], indexes[i*2 + 1]);
    }
    
    // we don't have so many points in the coalesced bucket, use sh mem processing
    bin=count;
    binCount=0;
    while(!__all_sync(0xFFFFFFFF, processed)) {
      if(!processed) {
        binCount=atomicAdd(&binCounts[bin], 1);
        if(binCount<7) {
          countsAndIndexes[bin*7*3 + binCount*3 + 0]=make_uint4(counts[0], indexes[0], counts[1], indexes[1]);
          countsAndIndexes[bin*7*3 + binCount*3 + 1]=make_uint4(counts[2], indexes[2], counts[3], indexes[3]);
          countsAndIndexes[bin*7*3 + binCount*3 + 2]=make_uint4(counts[4], indexes[4], counts[5], indexes[5]);
          buckets[bin*7 + binCount]=bucket;
          processed=true;
        }
      }
      if(binCount==6)
        writeIndex=atomicAdd(&histogram[1023-bin], 7);
      while(true) {
        mask=__ballot_sync(0xFFFFFFFF, binCount==6);
        if(mask==0)
          break;
        thread=31-__clz(mask);
        localBin=__shfl_sync(0xFFFFFFFF, bin, thread);
        localWriteIndex=__shfl_sync(0xFFFFFFFF, writeIndex, thread);
        if(warpThread<7) {
          localBucket=atomicExch(&buckets[localBin*7 + warpThread], 0xFFFFFFFF);
          while(localBucket==0xFFFFFFFF)
            localBucket=atomicExch(&buckets[localBin*7 + warpThread], 0xFFFFFFFF);
          sortedBuckets[localWriteIndex + warpThread]=localBucket;
        }
        __syncwarp(0xFFFFFFFF);
        if(warpThread<21) 
          sortedCountsAndIndexes[localWriteIndex*3 + warpThread]=countsAndIndexes[localBin*7*3 + warpThread];
        __syncwarp(0xFFFFFFFF);
        binCounts[localBin]=0;
        binCount=(thread==warpThread) ? 0 : binCount;
      }
    }
  }
  
  __syncthreads();

  for(int32_t i=warp;i<256;i+=warps) {
    binCount=binCounts[i];
    if(binCount>0) {
      if(warpThread==0) 
        writeIndex=atomicAdd(&histogram[1023-i], binCount);
      writeIndex=__shfl_sync(0xFFFFFFFF, writeIndex, 0);
      if(warpThread<binCount) 
        sortedBuckets[writeIndex + warpThread]=buckets[i*7 + warpThread];
      if(warpThread<binCount*3) 
        sortedCountsAndIndexes[writeIndex*3 + warpThread]=countsAndIndexes[i*7*3 + warpThread];
    }
  }
}
