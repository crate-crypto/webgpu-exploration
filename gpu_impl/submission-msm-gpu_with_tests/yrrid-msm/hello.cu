#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAddition(float* a, float* b, float* result, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        result[index] = a[index] + b[index];
    }
}

int main() {
    // Створюємо cudaEvent для вимірювання часу
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Запускаємо таймер
    hipEventRecord(start, 0);

    int size = 1000000;
    int byteSize = size * sizeof(float);

    // Задаємо вектори
    float* hostVectorA = new float[size];
    float* hostVectorB = new float[size];
    float* hostResult = new float[size];

    // Ініціалізуємо вектори
    for (int i = 0; i < size; ++i) {
        hostVectorA[i] = static_cast<float>(i);
        hostVectorB[i] = static_cast<float>(i * 2);
    }

    // Виділяємо пам'ять на пристрої (GPU)
    float* deviceVectorA, * deviceVectorB, * deviceResult;
    hipMalloc((void**)&deviceVectorA, byteSize);
    hipMalloc((void**)&deviceVectorB, byteSize);
    hipMalloc((void**)&deviceResult, byteSize);

    // Копіюємо дані з хоста на пристрій
    hipMemcpy(deviceVectorA, hostVectorA, byteSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceVectorB, hostVectorB, byteSize, hipMemcpyHostToDevice);

    // Задаємо конфігурацію запуску ядра
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    

    // Виклик ядра
    vectorAddition<<<gridSize, blockSize>>>(deviceVectorA, deviceVectorB, deviceResult, size);

    

    // Копіюємо результати обчислень з пристрою на хост
    hipMemcpy(hostResult, deviceResult, byteSize, hipMemcpyDeviceToHost);

    // Виводимо результати та час виконання
    // for (int i = 0; i < size; ++i) {
    //     std::cout << hostVectorA[i] << " + " << hostVectorB[i] << " = " << hostResult[i] << std::endl;
    // }
    

    // Вивільняємо пам'ять
    delete[] hostVectorA;
    delete[] hostVectorB;
    delete[] hostResult;
    hipFree(deviceVectorA);
    hipFree(deviceVectorB);
    hipFree(deviceResult);

    // Зупиняємо таймер
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Обчислюємо час виконання
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "Elapsed Time: " << elapsedTime << " ms" << std::endl;

    return 0;
}
